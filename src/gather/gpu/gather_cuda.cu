#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <stdio.h>

template<typename T>
__global__ void gather_kernel(T const *data, int64_t const* indices,
                            int indices_size, T *output, int output_size,
                            int pre_size, int axis_size, int post_size)
{
    int num_elem_per_thread = 2 + output_size / 6291456; // 针对本例大计算量，动态调整每个线程处理的元素个数
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x >= pre_size || y >= indices_size * post_size) {
        return;
    }

    for(int i = 0; i < num_elem_per_thread; i++) {
        const int idx = y * num_elem_per_thread + i;
        if(idx >= indices_size * post_size) {
            break;
        }

        const int indices_idx = idx / post_size;
        const int post_idx = idx % post_size;
        const int pre_idx = x;

        const int data_idx = pre_idx * axis_size * post_size + indices[indices_idx] * post_size + post_idx;
        const int output_idx = pre_idx * indices_size * post_size + idx;

        output[output_idx] = data[data_idx];
    }
}

extern "C" void gather(void const* data, int64_t const* indices,
                        int indices_size, void* output,  int output_size,
                        int pre_size, int axis_size, int post_size,
                        const int elem_size)
{
    int num_elem_per_thread = 2 + output_size / 6291456;
    // 将input都看成三部分，pre_size是axis前面的大小, axis_size是axis上的大小, post_size是axis后的大小
    // 根据gather的定义，output也可以相应看成三部分
    dim3 block_size(1, 128); // 该参数确保不超过block和grid上限
    dim3 grid_size((pre_size + block_size.x - 1) / block_size.x,
            (indices_size * post_size + block_size.y * num_elem_per_thread - 1) / (block_size.y * num_elem_per_thread));
    // x维度计算output的pre_size部分, y维度计算output的indices_size * post_size部分，通过计算x和y的索引可以得出input三个部分对应的索引
    if(elem_size == 2) {
        gather_kernel<half><<<grid_size, block_size>>>((half*)data, indices, indices_size, (half*)output, output_size, pre_size,
                            axis_size, post_size);
    } else if(elem_size == 4) {
        gather_kernel<float><<<grid_size, block_size>>>((float*)data, indices, indices_size, (float*)output, output_size, pre_size,
                            axis_size, post_size);
    }
    // cudaError_t error = cudaGetLastError();
    // if (error != cudaSuccess) {
    //     printf("CUDA error: %s\n", cudaGetErrorString(error));
    // }
}