
#include <hip/hip_runtime.h>
#include <cstdio>

static __global__ void gather(int unit, void *output, void const *indices, void const *input) {
}

extern "C" void gather_nv(int unit, int y, int x, int yi, int xi, int axis, void *output, void const *indices, void const *input) {
    printf("gather_nv called (%d, %d) -> (%d, %d), unit = %d, axis = %d\n", yi, xi, y, x, unit, axis);
    gather<<<1, 1>>>(unit, output, indices, input);
}
